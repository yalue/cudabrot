#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
extern "C" {
#include <SDL2/SDL.h>
}

// Controls the number of threads per block to use.
#define DEFAULT_BLOCK_SIZE (1024)

// Controls the default number of blocks to use.
#define DEFAULT_BLOCK_COUNT (16)

// This macro takes a hipError_t value and exits the program if it isn't equal
// to hipSuccess. (Calls the ErrorCheck function, defined later).
#define CheckCUDAError(val) (InternalCUDAErrorCheck((val), #val, __FILE__, __LINE__))

// This will be a magic value instructing the program to not explicitly set a
// CUDA device.
#define USE_DEFAULT_DEVICE (-1)

// The gamma value to use for gamma correction, or 1.0 if no gamma correction
// should be applied.
#define GAMMA_CORRECTION (1.1)

// The number of color channels in the resulting image. Should be 3 for RBG.
#define COLOR_CHANNELS (3)

// Holds the boundaries and sizes of the fractal, in both pixels and numbers
typedef struct {
  // The width and height of the image in pixels.
  int w;
  int h;
  // The boundaries of the fractal.
  double min_real;
  double min_imag;
  double max_real;
  double max_imag;
  // The distance between pixels in the real and imaginary axes.
  double delta_real;
  double delta_imag;
} FractalDimensions;

// This struct holds the parameters for different types of "iterations" needed
// when calculating the buddhabrot.
typedef struct {
  // Each CUDA thread in every block will sample this many random points.
  int samples_per_thread;
  // This is the maximum number of iterations to run to see if a point escapes.
  int max_escape_iterations;
  // If a point escapes in fewer than this many iterations, it will be ignored.
  int min_escape_iterations;
} IterationControl;

// Holds globals in a single namespace.
static struct {
  SDL_Window *window;
  SDL_Renderer *renderer;
  SDL_Texture *image;
  // The CUDA device to use. If this is -1, a device won't be set, which should
  // fall back to CUDA's normal device.
  int cuda_device;
  // This tracks the random number generator states for the GPU code.
  hiprandState_t *rng_states;
  // The number of threads and blocks to use when calculating the buddhabrot.
  int block_size, block_count;
  // The filename to which a bitmap image will be saved, or NULL if an image
  // should not be saved.
  char *output_image;
  // The number of iterations to check for escaping points in the buddhabrot.
  int buddhabrot_iterations;
  // The size and location of the fractal and output image.
  FractalDimensions dimensions;
  // The host and device buffers which contain the numbers of times an escaping
  // point's path crossed each point in the complex plane.
  uint32_t *device_buddhabrot;
  uint32_t *host_buddhabrot;
  // Buffers for the three different color channels, which will be calculated
  // separately and combined into the final image.
  uint8_t *color_channels[COLOR_CHANNELS];
} g;

// If any globals have been initialized, this will free them. (Relies on
// globals being set to 0 at the start of the program)
static void CleanupGlobals(void) {
  int i;
  if (g.renderer) SDL_DestroyRenderer(g.renderer);
  if (g.image) SDL_DestroyTexture(g.image);
  if (g.window) SDL_DestroyWindow(g.window);
  if (g.rng_states) hipFree(g.rng_states);
  if (g.device_buddhabrot) hipFree(g.device_buddhabrot);
  for (i = 0; i < COLOR_CHANNELS; i++) {
    if (g.color_channels[i]) free(g.color_channels[i]);
  }
  memset(&g, 0, sizeof(g));
}

// Returns the current time in seconds.
static double CurrentSeconds(void) {
  struct timespec ts;
  if (clock_gettime(CLOCK_REALTIME, &ts) != 0) {
    printf("Error getting time.\n");
    exit(1);
  }
  return ((double) ts.tv_sec) + (((double) ts.tv_nsec) / 1e9);
}

// Prints an error message and exits the program if the hipError_t value is
// not equal to hipSuccess. Generally, this will be called via the
// CheckCudaError macro.
static void InternalCUDAErrorCheck(hipError_t result, const char *fn,
    const char *file, int line) {
  if (result == hipSuccess) return;
  printf("CUDA error %d in %s, line %d (%s)\n", (int) result, file, line, fn);
  CleanupGlobals();
  exit(1);
}

// Sets up the SDL window and resources. Must be called after g.w and g.h have
// been set.
static void SetupSDL(void) {
  if (SDL_Init(SDL_INIT_EVERYTHING) < 0) {
    printf("SDL error %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.window = SDL_CreateWindow("Rendered image", SDL_WINDOWPOS_UNDEFINED,
    SDL_WINDOWPOS_UNDEFINED, g.dimensions.w, g.dimensions.h, SDL_WINDOW_SHOWN |
    SDL_WINDOW_RESIZABLE);
  if (!g.window) {
    printf("Error creating SDL window: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.renderer = SDL_CreateRenderer(g.window, -1, SDL_RENDERER_ACCELERATED);
  if (!g.renderer) {
    printf("Error creating SDL renderer: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.image = SDL_CreateTexture(g.renderer, SDL_PIXELFORMAT_RGBA8888,
    SDL_TEXTUREACCESS_STREAMING, g.dimensions.w, g.dimensions.h);
  if (!g.image) {
    printf("Failed creating SDL texture: %s\n", SDL_GetError());
    exit(1);
  }
}

// This function is used to initialize the RNG states to use when generating
// starting points in the buddhabrot calculation. The states array must hold
// one entry for every thread in every block.
__global__ void InitializeRNG(uint64_t seed, hiprandState_t *states) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  hiprand_init(seed, index, 0, states + index);
}

// Allocates CUDA memory and calculates block/grid sizes. Must be called after
// g.w and g.h have been set.
static void SetupCUDA(void) {
  int i;
  if (g.cuda_device != USE_DEFAULT_DEVICE) {
    CheckCUDAError(hipSetDevice(g.cuda_device));
  }
  size_t buffer_size = g.dimensions.w * g.dimensions.h;

  // Initialize the host and device image buffers.
  CheckCUDAError(hipMalloc(&(g.device_buddhabrot), buffer_size *
    sizeof(uint32_t)));
  CheckCUDAError(hipMemset(g.device_buddhabrot, 0, buffer_size *
    sizeof(uint32_t)));
  g.host_buddhabrot = (uint32_t *) malloc(buffer_size * sizeof(uint32_t));
  if (!g.host_buddhabrot) {
    printf("Failed allocating host buddhabrot buffer.\n");
    CleanupGlobals();
    exit(1);
  }
  memset(g.host_buddhabrot, 0, buffer_size * sizeof(uint32_t));

  // Initialize the RNG state for the device.
  CheckCUDAError(hipMalloc(&(g.rng_states), g.block_size * g.block_count *
    sizeof(hiprandState_t)));
  InitializeRNG<<<g.block_size, g.block_count>>>(1337, g.rng_states);
  CheckCUDAError(hipDeviceSynchronize());

  // Allocate the color channels for the combined image.
  for (i = 0; i < COLOR_CHANNELS; i++) {
    g.color_channels[i] = (uint8_t *) malloc(buffer_size);
    if (!g.color_channels[i]) {
      printf("Failed allocating color channel %d buffer.\n", i);
      CleanupGlobals();
      exit(1);
    }
    memset(g.color_channels[i], 0, buffer_size);
  }
}

// This should be used to update the pixel data for a point that is encountered
// in the set.
__device__ void IncrementPixelCounter(int row, int col, uint32_t *data,
    FractalDimensions *d) {
  int r, c;
  r = row;
  c = col;
  if ((r >= 0) && (r < d->h) && (c >= 0) && (c < d->h)) {
    data[r * d->w + c] += 4;
  }
}

// This kernel takes a list of points which escape the mandelbrot set, and, for
// each iteration of the point, increments its location in the data array.
__global__ void DrawBuddhabrot(FractalDimensions dimensions, uint32_t *data,
    IterationControl iterations, hiprandState_t *states) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  hiprandState_t *rng = states + index;
  int i, j, point_escaped, record_path, row, col;
  float start_real, start_imag, current_real, current_imag, tmp;
  float real_range = dimensions.max_real - dimensions.min_real;
  float imag_range = dimensions.max_imag - dimensions.min_imag;
  record_path = 0;
  point_escaped = 1;
  for (i = 0; i < iterations.samples_per_thread; i++) {
    // Calculate a new starting point only if the previous point didn't escape.
    // Otherwise, we'll use the same starting point, and record the point's
    // path.
    if (!record_path) {
      start_real = hiprand_uniform(rng) * real_range + dimensions.min_real;
      start_imag = hiprand_uniform(rng) * imag_range + dimensions.min_imag;
    }
    point_escaped = 0;
    current_real = start_real;
    current_imag = start_imag;
    for (j = 0; j < iterations.max_escape_iterations; j++) {
      tmp = (current_real * current_real) - (current_imag * current_imag) +
        start_real;
      current_imag = 2 * current_real * current_imag + start_imag;
      current_real = tmp;
      row = (current_imag - dimensions.min_imag) / dimensions.delta_imag;
      col = (current_real - dimensions.min_real) / dimensions.delta_real;
      if (record_path) {
        IncrementPixelCounter(row, col, data, &dimensions);
      }
      // If the point escapes, stop iterating and indicate the loop ended due
      // to the point escaping.
      if (((current_real * current_real) + (current_imag * current_imag)) >
        4) {
        point_escaped = 1;
        break;
      }
    }
    // Record the next path if the point didn't escape and we weren't already
    // recording.
    if (point_escaped && !record_path) {
      // Enables ignoring paths that escape too quickly.
      if (j > iterations.min_escape_iterations) record_path = 1;
    } else {
      record_path = 0;
    }
  }
}

static uint8_t Clamp(double v) {
  if (v <= 0) return 0;
  if (v >= 255) return 255;
  return (uint8_t) v;
}

// Returns the amount to multiply the original count by in order to get a value
// by which buddhabrot counts can be multiplied to get a number between 0 and
// 255.
static double GetLinearColorScale(void) {
  int x, y, index;
  uint32_t max = 0;
  index = 0;
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      if (g.host_buddhabrot[index] > max) max = g.host_buddhabrot[index];
    }
  }
  return 255.0 / ((double) max);
}

// Returns the gamma-corrected 8-bit color channel value given a buddhabrot
// iteration count c.
static uint8_t DoGammaCorrection(uint32_t c, double linear_scale) {
  double scaled = ((double) c) * linear_scale;
  scaled = 255 * log(c + 1) / log(255);
  return Clamp(255 * pow(scaled / 255, 1 / GAMMA_CORRECTION));
}

// Fills in a single color channel from the current host_buddhabrot buffer.
static void SetColorChannel(uint8_t *color) {
  int x, y;
  uint8_t color_value;
  double linear_scale = GetLinearColorScale();
  uint32_t *host_data = g.host_buddhabrot;
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      color_value = DoGammaCorrection(*host_data, linear_scale);
      *color = color_value;
      color++;
      host_data++;
    }
  }
}

// Renders the fractal image.
static void RenderImage(void) {
  int channel;
  size_t data_size = g.dimensions.w * g.dimensions.h;
  double seconds;
  IterationControl iterations;
  iterations.min_escape_iterations = 20;
  iterations.samples_per_thread = 100;
  iterations.max_escape_iterations = g.buddhabrot_iterations;

  for (channel = 0; channel < COLOR_CHANNELS; channel++) {
    printf("Calculating color channel %d.\n", channel);
    printf("Calculating buddhabrot.\n");
    seconds = CurrentSeconds();
    DrawBuddhabrot<<<g.block_count, g.block_size>>>(g.dimensions,
       g.device_buddhabrot, iterations, g.rng_states);
    CheckCUDAError(hipGetLastError());
    CheckCUDAError(hipMemcpy(g.host_buddhabrot, g.device_buddhabrot,
      data_size * sizeof(uint32_t), hipMemcpyDeviceToHost));
    printf("  Buddhabrot took %f seconds.\n", CurrentSeconds() - seconds);

    SetColorChannel(g.color_channels[channel]);
    // Color channels will only differ by a fixed number of iterations for now.
    iterations.max_escape_iterations /= 10;
    iterations.min_escape_iterations /= 2;
  }
}

// Copies data from the host-side data buffer to the texture drawn to the SDL
// window.
static void UpdateDisplayedImage(void) {
  int x, y;
  uint8_t *image_pixels;
  int image_pitch, to_skip_per_row, pixel_number;
  if (SDL_LockTexture(g.image, NULL, (void **) (&image_pixels), &image_pitch)
    < 0) {
    printf("Error locking SDL texture: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  // Abide by the image pitch, and skip unaffected bytes in each row.
  // (image_pitch should usually be equal to g.w * 4 anyway).
  to_skip_per_row = image_pitch - (g.dimensions.w * 4);
  pixel_number = 0;
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      // The byte order is ABGR
      image_pixels[0] = 0xff;
      image_pixels[1] = g.color_channels[2][pixel_number];
      image_pixels[2] = g.color_channels[1][pixel_number];
      image_pixels[3] = g.color_channels[0][pixel_number];
      image_pixels += 4;
      pixel_number++;
    }
    image_pixels += to_skip_per_row;
  }
  SDL_UnlockTexture(g.image);
}

// Runs the main loop to display the SDL window. This will return when SDL
// detects an exit event.
static void SDLWindowLoop(void) {
  SDL_Event event;
  int quit = 0;
  // Update the display once every 30 ms (not really necessary for now, while
  // it doesn't change...
  while (!quit) {
    while (SDL_PollEvent(&event)) {
      if (event.type == SDL_QUIT) {
        quit = 1;
        break;
      }
    }
    UpdateDisplayedImage();
    if (SDL_RenderCopy(g.renderer, g.image, NULL, NULL) < 0) {
      printf("Error rendering image: %s\n", SDL_GetError());
      CleanupGlobals();
      exit(1);
    }
    SDL_RenderPresent(g.renderer);
    usleep(20000);
  }
}

// Sets the resolution, scaling the complex boundaries to maintain an aspect
// ratio.
static void SetResolution(int width, int height) {
  FractalDimensions *dims = &(g.dimensions);
  double ratio = ((double) height) / ((double) width);
  // The horizontal width for which the complex plane is shown.
  double real_width = 4.0;
  double imag_width = real_width * ratio;
  dims->w = width;
  dims->h = height;
  dims->min_real = -(real_width / 2.0);
  dims->max_real = dims->min_real + real_width;
  dims->min_imag = -(imag_width / 2.0);
  dims->max_imag = dims->min_imag + imag_width;
  dims->delta_imag = imag_width / ((double) height);
  dims->delta_real = real_width / ((double) width);
}

// If a filename has been set for saving the image, this will attempt to save
// the image to the file.
static void SaveImage(void) {
  void *pixel_data;
  SDL_Surface *image_surface = NULL;
  int w = g.dimensions.w;
  int h = g.dimensions.h;
  // Don't do anything if the output filename wasn't set.
  if (!g.output_image) return;

  // In SDL 2, we need to copy the image from the renderer and create an SDL
  // surface in order to save a bitmap.
  pixel_data = malloc(w * h * 4);
  if (!pixel_data) {
    printf("Failed allocating space to save an image.\n");
    CleanupGlobals();
    exit(1);
  }
  if (SDL_RenderReadPixels(g.renderer, NULL, SDL_PIXELFORMAT_RGBA8888,
    pixel_data, w * 4) != 0) {
    printf("Failed getting BMP image data: %s\n", SDL_GetError());
    free(pixel_data);
    CleanupGlobals();
    exit(1);
  }
  image_surface = SDL_CreateRGBSurfaceFrom(pixel_data, w, h, 32, w * 4, 0xff,
    0xff00, 0xff0000, 0xff000000);
  if (!image_surface) {
    printf("Failed creating BMP surface: %s\n", SDL_GetError());
    free(pixel_data);
    CleanupGlobals();
    exit(1);
  }
  if (SDL_SaveBMP(image_surface, g.output_image) != 0) {
    printf("Failed saving BMP file: %s\n", SDL_GetError());
    SDL_FreeSurface(image_surface);
    free(pixel_data);
    CleanupGlobals();
    exit(1);
  }
  printf("Successfully saved %s\n", g.output_image);
  SDL_FreeSurface(image_surface);
  free(pixel_data);
}

static void PrintUsage(char *program_name) {
  printf("Usage: %s [options]\n\n", program_name);
  printf("Options may be one or more of the following:\n"
    "  --help: Prints these instructions.\n"
    "  -d <CUDA device number>: Can be used to set which GPU to use.\n"
    "     Defaults to the default GPU.\n"
    "  -s <output file name>: If provided, the rendered image will be saved\n"
    "     to a bitmap file with the given name, in addition to being\n"
    "     displayed in a window.\n"
    "  -b <buddhabrot iterations>: The number of iterations to use for the\n"
    "     buddhabrot calculation. Defaults to 1000.\n");
  exit(0);
}

// Returns an integer at the argument after index in argv. Exits if the integer
// is invalid.
static int ParseIntArg(int argc, char **argv, int index) {
  char *tmp = NULL;
  int to_return = 0;
  if ((index + 1) >= argc) {
    printf("Argument %s needs a value.\n", argv[index]);
    PrintUsage(argv[0]);
  }
  to_return = strtol(argv[index + 1], &tmp, 10);
  if (*tmp != 0) {
    printf("Invalid number given to argument %s: %s\n", argv[index],
      argv[index + 1]);
    PrintUsage(argv[0]);
  }
  return to_return;
}

// Processes command-line arguments, setting values in the globals struct as
// necessary.
static void ParseArguments(int argc, char **argv) {
  for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "--help") == 0) {
      PrintUsage(argv[0]);
    }
    if (strcmp(argv[i], "-d") == 0) {
      g.cuda_device = ParseIntArg(argc, argv, i);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-s") == 0) {
      if ((i + 1) >= argc) {
        printf("Missing output file name.\n");
        PrintUsage(argv[0]);
      }
      i++;
      g.output_image = argv[i];
      continue;
    }
    if (strcmp(argv[i], "-b") == 0) {
      g.buddhabrot_iterations = ParseIntArg(argc, argv, i);
      i++;
      continue;
    }
    // Unrecognized argument, print the usage string.
    printf("Invalid argument: %s\n", argv[i]);
    PrintUsage(argv[0]);
  }
}

int main(int argc, char **argv) {
  memset(&g, 0, sizeof(g));
  g.buddhabrot_iterations = 1000;
  g.block_size = DEFAULT_BLOCK_SIZE;
  g.block_count = DEFAULT_BLOCK_COUNT;
  SetResolution(1000, 1000);
  g.cuda_device = USE_DEFAULT_DEVICE;
  ParseArguments(argc, argv);
  printf("Calculating image...\n");
  SetupCUDA();
  RenderImage();
  printf("Done!\n");
  SetupSDL();
  SDLWindowLoop();
  SaveImage();
  CleanupGlobals();
  return 0;
}
