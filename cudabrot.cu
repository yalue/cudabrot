#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <signal.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

// Controls the number of threads per block to use.
#define DEFAULT_BLOCK_SIZE (512)

// Controls the default number of blocks to use.
#define DEFAULT_BLOCK_COUNT (64)

// The name given to the output file if one isn't specified.
#define DEFAULT_OUTPUT_NAME "output.pgm"

// This macro takes a cudaError_t value and exits the program if it isn't equal
// to cudaSuccess. (Calls the ErrorCheck function, defined later).
#define CheckCUDAError(val) (InternalCUDAErrorCheck((val), #val, __FILE__, __LINE__))

// Increasing this may increase efficiency, but decrease responsiveness to
// signals.
#define SAMPLES_PER_THREAD (50)

// If the number of max iterations exceeds this value, the samples per thread
// will be reduced to 1 maintain responsiveness.
#define SAMPLE_REDUCTION_THRESHOLD (60000)

// The RNG seed used when initializing the RNG states on the GPU.
#define DEFAULT_RNG_SEED (1337)

// Holds the boundaries and sizes of the fractal, in both pixels and numbers
typedef struct {
  // The width and height of the image in pixels.
  int w;
  int h;
  // The boundaries of the fractal.
  double min_real;
  double min_imag;
  double max_real;
  double max_imag;
  // The distance between pixels in the real and imaginary axes.
  double delta_real;
  double delta_imag;
} FractalDimensions;

// This struct holds the parameters for different types of "iterations" needed
// when calculating the Buddhabrot.
typedef struct {
  // Each CUDA thread in every block will sample this many random points.
  int samples_per_thread;
  // This is the maximum number of iterations to run to see if a point escapes.
  int max_escape_iterations;
  // If a point escapes in fewer than this many iterations, it will be ignored.
  int min_escape_iterations;
} IterationControl;

// Holds global state in a single struct.
static struct {
  // The CUDA device to use. If this is -1, a device won't be set, which should
  // fall back to CUDA's normal device.
  int cuda_device;
  // This tracks the random number generator states for the GPU code.
  hiprandState_t *rng_states;
  // The number of threads and blocks to use when calculating the Buddhabrot.
  int block_size, block_count;
  // The filename to which a bitmap image will be saved, or NULL if an image
  // should not be saved.
  const char *output_image;
  // The number of seconds to run the calculation. If negative, wait for a
  // signal instead.
  double seconds_to_run;
  // If this is nonzero, the program should save the image and quit as soon as
  // the current iteration finishes.
  int quit_signal_received;
  // Holds various iteration-related settings.
  IterationControl iterations;
  // The size and location of the fractal and output image.
  FractalDimensions dimensions;
  // The host and device buffers which contain the numbers of times an escaping
  // point's path crossed each point in the complex plane.
  uint64_t *device_buddhabrot;
  uint64_t *host_buddhabrot;
  // The gamma value for gamma correction.
  double gamma_correction;
  // Buffer for a single grayscale image.
  uint16_t *grayscale_image;
} g;

// If any globals have been initialized, this will free them. (Relies on
// globals being set to 0 at the start of the program)
static void CleanupGlobals(void) {
  if (g.rng_states) hipFree(g.rng_states);
  if (g.device_buddhabrot) hipFree(g.device_buddhabrot);
  if (g.rng_states) hipFree(g.rng_states);
  if (g.grayscale_image) free(g.grayscale_image);
  memset(&g, 0, sizeof(g));
}

// Returns the current time in seconds.
static double CurrentSeconds(void) {
  struct timespec ts;
  if (clock_gettime(CLOCK_REALTIME, &ts) != 0) {
    printf("Error getting time.\n");
    exit(1);
  }
  return ((double) ts.tv_sec) + (((double) ts.tv_nsec) / 1e9);
}

// Prints an error message and exits the program if the cudaError_t value is
// not equal to cudaSuccess. Generally, this will be called via the
// CheckCUDAError macro.
static void InternalCUDAErrorCheck(hipError_t result, const char *fn,
    const char *file, int line) {
  if (result == hipSuccess) return;
  printf("CUDA error %d in %s, line %d (%s)\n", (int) result, file, line, fn);
  CleanupGlobals();
  exit(1);
}

// This function is used to initialize the RNG states to use when generating
// starting points in the Buddhabrot calculation. The states array must hold
// one entry for every thread in every block.
__global__ void InitializeRNG(uint64_t seed, hiprandState_t *states) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  hiprand_init(seed, index, 0, states + index);
}

// Allocates CUDA memory and calculates block/grid sizes. Must be called after
// g.w and g.h have been set.
static void SetupCUDA(void) {
  float gpu_memory_needed, cpu_memory_needed;
  CheckCUDAError(hipSetDevice(g.cuda_device));
  size_t buffer_size = g.dimensions.w * g.dimensions.h;
  // The GPU will need space for the image and the RNG states.
  gpu_memory_needed = buffer_size * sizeof(uint64_t) +
    (g.block_size * g.block_count * sizeof(hiprandState_t));
  gpu_memory_needed /= (1024.0 * 1024.0);
  // The CPU needs space for the image and grayscale conversion.
  cpu_memory_needed = buffer_size * sizeof(uint64_t) +
    buffer_size * sizeof(uint16_t);
  cpu_memory_needed /= (1024.0 * 1024.0);
  printf("Approximate memory needed: %.03f MiB GPU, %.03f MiB CPU\n",
    gpu_memory_needed, cpu_memory_needed);

  // Initialize the host and device image buffers.
  CheckCUDAError(hipMalloc(&(g.device_buddhabrot), buffer_size *
    sizeof(uint64_t)));
  CheckCUDAError(hipMemset(g.device_buddhabrot, 0, buffer_size *
    sizeof(uint64_t)));
  g.host_buddhabrot = (uint64_t *) malloc(buffer_size * sizeof(uint64_t));
  if (!g.host_buddhabrot) {
    printf("Failed allocating host Buddhabrot buffer.\n");
    CleanupGlobals();
    exit(1);
  }
  memset(g.host_buddhabrot, 0, buffer_size * sizeof(uint64_t));

  // Initialize the RNG state for the device.
  CheckCUDAError(hipMalloc(&(g.rng_states), g.block_size * g.block_count *
    sizeof(hiprandState_t)));
  InitializeRNG<<<g.block_size, g.block_count>>>(DEFAULT_RNG_SEED,
    g.rng_states);
  CheckCUDAError(hipDeviceSynchronize());

  g.grayscale_image = (uint16_t *) malloc(buffer_size * sizeof(uint16_t));
  if (!g.grayscale_image) {
    printf("Failed allocating grayscale image.\n");
    CleanupGlobals();
    exit(1);
  }
  memset(g.grayscale_image, 0, buffer_size * sizeof(uint16_t));
}

// This returns nonzero if the given point is in the main cardioid of the set
// and is therefore guaranteed to not escape.
inline __device__ int InMainCardioid(double real, double imag) {
  // This algorithm was taken from the Wikipedia Mandelbrot set page.
  double imag_squared = imag * imag;
  double q = (real - 0.25);
  q = q * q + imag_squared;
  return q * (q + (real - 0.25)) < (imag_squared * 0.25);
}

// This returns nonzero if the given point is in the order 2 bulb of the set
// and therefore guaranteed to not escape.
inline __device__ int InOrder2Bulb(double real, double imag) {
  double tmp = real + 1;
  tmp = tmp * tmp;
  return (tmp + (imag * imag)) < (1.0 / 16.0);
}

// This should be used to update the pixel data for a point that is encountered
// in the set.
inline __device__ void IncrementPixelCounter(double real, double imag,
    uint64_t *data, FractalDimensions *d) {
  int row, col;
  // There's a small issue here with integer-dividing where values that should
  // be immediately outside of the canvas can still appear on row or col 0, so
  // just return early if we're outside the boundary.
  if ((real < d->min_real) || (imag < d->min_imag)) return;
  col = (real - d->min_real) / d->delta_real;
  row = (imag - d->min_imag) / d->delta_imag;
  if ((row >= 0) && (row < d->h) && (col >= 0) && (col < d->w)) {
    data[(row * d->w) + col] += 1;
  }
}

// Does the Mandelbrot-set iterations for the given (real, imag) point. Returns
// the number of iterations before the point escapes, or max_iterations if the
// point never escapes.
inline __device__ int IterateMandelbrot(double start_real, double start_imag,
    int max_iterations) {
  double tmp, real, imag;
  int i;
  real = start_real;
  imag = start_imag;
  for (i = 0; i < max_iterations; i++) {
    tmp = (real * real) - (imag * imag) + start_real;
    imag = 2 * real * imag + start_imag;
    real = tmp;
    // If the point escapes, stop iterating and indicate the loop ended due
    // to the point escaping.
    if (((real * real) + (imag * imag)) > 4) return i;
  }
  // The point didn't escape, return max_iterations.
  return max_iterations;
}

// Like IterateMandelbrot, but records the point's path. For efficiency, this
// function also has an important difference from IterateMandelbrot: *it does
// not check the max iterations*. This is important! Do not call this function
// for a point unless you're sure that it escapes in a finite number of
// iterations.
inline __device__ void IterateAndRecord(double start_real, double start_imag,
    uint64_t *data, FractalDimensions *d) {
  double tmp, real, imag;
  real = start_real;
  imag = start_imag;
  while (1) {
    tmp = (real * real) - (imag * imag) + start_real;
    imag = 2 * real * imag + start_imag;
    real = tmp;
    IncrementPixelCounter(real, imag, data, d);
    // Stop iterating when the point escapes. This must be *guaranteed* to
    // happen by the caller performing a prior check!
    if (((real * real) + (imag * imag)) > 4) break;
  }
}


// This kernel is responsible for drawing the paths of "particles" that escape
// the mandelbrot set. It works as follows:
//
// 1. For each "sample", compute a new random starting point in the complex
//    plane
// 2. Do the normal mandelbrot iterations on the starting point, *without*
//    recording its path
// 3. If the point didn't escape the path, take a new sample (return to step 1)
// 4. If the point escaped (within the min and max iteration limits), then
//    repeat the mandelbrot iterations (e.g. step 2), except record its path
//    by incrementing the pixel value for every point it passes through.
__global__ void DrawBuddhabrot(FractalDimensions dimensions, uint64_t *data,
    IterationControl iterations, hiprandState_t *states) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  hiprandState_t *rng = states + index;
  int sample, iterations_needed, max_iterations, min_iterations;
  double real, imag;
  max_iterations = iterations.max_escape_iterations;
  min_iterations = iterations.min_escape_iterations;

  // We're going to pick a number of random starting points configured by the
  // iterations.samples_per_thread setting.
  for (sample = 0; sample < iterations.samples_per_thread; sample++) {
    // Sample across the entire domain of the set regardless of our "canvas"
    real = (hiprand_uniform_double(rng) * 4.0) - 2.0;
    imag = (hiprand_uniform_double(rng) * 4.0) - 2.0;

    // Optimization: we know ahead of time that points from the main cardioid
    // and the largest "bulb" will never escape, and it's fast to check them.
    if (InMainCardioid(real, imag) || InOrder2Bulb(real, imag)) continue;

    // Now, do the normal Mandelbrot iterations to see how quickly the point
    // escapes (if it does). However, we won't record the path yet.
    iterations_needed = IterateMandelbrot(real, imag, max_iterations);

    // Don't record the path if the point never escaped, or if it escaped too
    // quickly.
    if (iterations_needed >= max_iterations) continue;
    if (iterations_needed < min_iterations) continue;

    // At this point, do the Mandelbrot iterations, but actually record the
    // path because we know the point is "good".
    IterateAndRecord(real, imag, data, &dimensions);
  }
}

static uint16_t Clamp(double v) {
  if (v <= 0) return 0;
  if (v >= 0xffff) return 0xffff;
  return (uint16_t) v;
}

// Returns the amount to multiply the original count by in order to get a value
// by which Buddhabrot counts can be multiplied to get a number between 0 and
// 0xffff.
static double GetLinearColorScale(void) {
  int x, y, index;
  uint64_t max = 0;
  double to_return;
  index = 0;
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      if (g.host_buddhabrot[index] > max) max = g.host_buddhabrot[index];
      index++;
    }
  }
  to_return = ((double) 0xffff) / ((double) max);
  printf("Max value: %lu, scale: %f\n", (unsigned long) max, to_return);
  return to_return;
}

// Returns the gamma-corrected 16-bit color channel value given a Buddhabrot
// iteration count c.
static uint16_t DoGammaCorrection(uint64_t c, double linear_scale) {
  double max = 0xffff;
  double scaled = ((double) c) * linear_scale;
  // Don't do gamma correction if the gamma correction argument was negative.
  if (g.gamma_correction <= 0.0) return scaled;
  return Clamp(max * pow(scaled / max, 1 / g.gamma_correction));
}

// Converts the buffer of 64-bit pixel values to a gamma-corrected grayscale
// image with 16-bit colors. The 64-bit values are scaled to fill the 16-bit
// color range.
static void SetGrayscalePixels(void) {
  int x, y;
  uint16_t color_value;
  double linear_scale = GetLinearColorScale();
  uint64_t *host_data = g.host_buddhabrot;
  uint16_t *grayscale = g.grayscale_image;
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      color_value = DoGammaCorrection(*host_data, linear_scale);
      *grayscale = color_value;
      grayscale++;
      host_data++;
    }
  }
}

// Renders the fractal image.
static void RenderImage(void) {
  int passes_count = 0;
  size_t data_size = g.dimensions.w * g.dimensions.h;
  double start_seconds;
  printf("Calculating Buddhabrot.\n");
  if (g.seconds_to_run < 0) {
    printf("Press ctrl+C to finish.\n");
  } else {
    printf("Running for %.03f seconds.\n", g.seconds_to_run);
  }

  // Run until either the time elapsed or we've received a SIGINT.
  start_seconds = CurrentSeconds();
  while (!g.quit_signal_received) {
    passes_count++;
    DrawBuddhabrot<<<g.block_count, g.block_size>>>(g.dimensions,
      g.device_buddhabrot, g.iterations, g.rng_states);
    CheckCUDAError(hipDeviceSynchronize());
    if ((g.seconds_to_run >= 0) && ((CurrentSeconds() - start_seconds) >
      g.seconds_to_run)) {
      break;
    }
  }

  // Copy the resulting image to CPU memory, and convert the pixels to proper
  // grayscale values.
  CheckCUDAError(hipMemcpy(g.host_buddhabrot, g.device_buddhabrot,
    data_size * sizeof(uint64_t), hipMemcpyDeviceToHost));
  printf("%d Buddhabrot passes took %f seconds.\n", passes_count,
    CurrentSeconds() - start_seconds);
  SetGrayscalePixels();
}

// Recomputes the spacing between pixels in the image. Returns 0 if any image-
// dimension setting is invalid. Otherwise, returns 1.
static int RecomputePixelDeltas(void) {
  FractalDimensions *dims = &(g.dimensions);
  if (dims->w <= 0) {
    printf("Output width must be positive.\n");
    return 0;
  }
  if (dims->h <= 0) {
    printf("Output height must be positive.\n");
    return 0;
  }
  if (dims->max_real <= dims->min_real) {
    printf("Maximum real value must be greater than minimum real value.\n");
    return 0;
  }
  if (dims->max_imag <= dims->min_imag) {
    printf("Minimum imaginary value must be greater than maximum imaginary "
      "value.\n");
    return 0;
  }
  dims->delta_imag = (dims->max_imag - dims->min_imag) / ((double) dims->h);
  dims->delta_real = (dims->max_real - dims->min_real) / ((double) dims->w);
  return 1;
}

// Sets the image boundaries and dimensions to their default values.
static void SetDefaultCanvas(void) {
  FractalDimensions *dims = &(g.dimensions);
  memset(dims, 0, sizeof(*dims));
  dims->w = 1000;
  dims->h = 1000;
  dims->min_real = -2.0;
  dims->max_real = 2.0;
  dims->min_imag = -2.0;
  dims->max_imag = 2.0;
  if (!RecomputePixelDeltas()) {
    printf("Internal error setting default canvas boundaries!\n");
    exit(1);
  }
}

// If a filename has been set for saving the image, this will attempt to save
// the image to the file. This can modify the image buffer! (For changing byte
// order.)
static void SaveImage(void) {
  uint16_t tmp;
  int i;
  int pixel_count = g.dimensions.w * g.dimensions.h;
  FILE *output = fopen(g.output_image, "wb");
  if (!output) {
    printf("Failed opening output image.\n");
    return;
  }
  if (fprintf(output, "P5\n%d %d\n%d\n", g.dimensions.w, g.dimensions.h,
    0xffff) <= 0) {
    printf("Failed writing pgm header.\n");
    fclose(output);
    return;
  }
  // Flip the byte-order for the image. This assumes the program is running on
  // a little-endian architecture. I'll fix it if there's ever a demand to run
  // this on something other than Linux on x86 or ARM64 (lol).
  for (i = 0; i < pixel_count; i++) {
    tmp = g.grayscale_image[i];
    tmp = ((tmp & 0xff) << 8) | (tmp >> 8);
    g.grayscale_image[i] = tmp;
  }
  if (!fwrite(g.grayscale_image, pixel_count * sizeof(uint16_t), 1, output)) {
    printf("Failed writing pixel data.\n");
    fclose(output);
    return;
  }
  fclose(output);
}

static void PrintUsage(char *program_name) {
  printf("Usage: %s [options]\n\n", program_name);
  printf("Options may be one or more of the following:\n"
    "  --help: Prints these instructions.\n"
    "  -d <device number>: Sets which GPU to use. Defaults to GPU 0.\n"
    "  -o <output file name>: If provided, the rendered image will be saved\n"
    "     to a bitmap file with the given name. Otherwise, saves the image\n"
    "     to " DEFAULT_OUTPUT_NAME ".\n"
    "  -m <max escape iterations>: The maximum number of iterations to use\n"
    "     before giving up on seeing whether a point escapes.\n"
    "  -c <min escape iterations>: If a point escapes before this number of\n"
    "     iterations, it will be ignored.\n"
    "  -g <gamma correction>: A gamma-correction value to use on the\n"
    "     resulting image. If negative, no gamma correction will occur.\n"
    "  -t <seconds to run>: A number of seconds to run the calculation for.\n"
    "     Defaults to 10.0. If negative, the program will run continuously\n"
    "     and will terminate (saving the image) when it receives a SIGINT.\n"
    "  -w <width>: The width of the output image, in pixels. Defaults to\n"
    "     1000.\n"
    "  -h <height>: The height of the output image, in pixels. Defaults to\n"
    "     1000.\n"
    "\n"
    "The following settings control the location of the output image on the\n"
    "complex plane, but samples are always drawn from the entire Mandelbrot-\n"
    "set domain (-2-2i to 2+2i). So these settings can be used to save\n"
    "memory or \"crop\" the output, but won't otherwise speed up rendering:\n"
    "  --min-real <min real>: The minimum value along the real axis to\n"
    "             include in the output image. Defaults to -2.0.\n"
    "  --max-real <max real>: The maximum value along the real axis to\n"
    "             include in the output image. Defaults to 2.0.\n"
    "  --min-imag <min imag>: The minimum value along the imaginary axis to\n"
    "             include in the output image. Defaults to -2.0.\n"
    "  --max-imag <max imag>: The maximum value along the imaginary axis to\n"
    "             include in the output image. Defaults to 2.0.\n"
    "");
  exit(0);
}

// Returns an integer at the argument after index in argv. Exits if the integer
// is invalid. Takes the index before the expected int value in order to print
// better error messages.
static int ParseIntArg(int argc, char **argv, int index) {
  char *tmp = NULL;
  int to_return = 0;
  if ((index + 1) >= argc) {
    printf("Argument %s needs a value.\n", argv[index]);
    PrintUsage(argv[0]);
  }
  to_return = strtol(argv[index + 1], &tmp, 10);
  // Make sure that, if tmp is a null character, that the argument wasn't
  // simply a string with no content.
  if ((*tmp != 0) || (argv[index + 1][0] == 0)) {
    printf("Invalid number given to argument %s: %s\n", argv[index],
      argv[index + 1]);
    PrintUsage(argv[0]);
  }
  return to_return;
}

// Like ParseIntArg, except expects a floating-point double arg.
static double ParseDoubleArg(int argc, char **argv, int index) {
  char *tmp = NULL;
  double to_return = 0.0;
  if ((index + 1) >= argc) {
    printf("Argument %s needs a value.\n", argv[index]);
    PrintUsage(argv[0]);
  }
  to_return = strtod(argv[index + 1], &tmp);
  if ((*tmp != 0) || (argv[index + 1][0] == 0)) {
    printf("Invalid number given to argument %s: %s\n", argv[index],
      argv[index + 1]);
    PrintUsage(argv[0]);
  }
  return to_return;
}

// Processes command-line arguments, setting values in the globals struct as
// necessary.
static void ParseArguments(int argc, char **argv) {
  for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "--help") == 0) {
      PrintUsage(argv[0]);
    }
    if (strcmp(argv[i], "-d") == 0) {
      g.cuda_device = ParseIntArg(argc, argv, i);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-o") == 0) {
      if ((i + 1) >= argc) {
        printf("Missing output file name.\n");
        PrintUsage(argv[0]);
      }
      i++;
      g.output_image = argv[i];
      continue;
    }
    if (strcmp(argv[i], "-m") == 0) {
      g.iterations.max_escape_iterations = ParseIntArg(argc, argv, i);
      if (g.iterations.max_escape_iterations > SAMPLE_REDUCTION_THRESHOLD) {
        // Maintain responsiveness with a huge number of iterations by reducing
        // the samples per thread.
        g.iterations.samples_per_thread = 1;
      }
      i++;
      continue;
    }
    if (strcmp(argv[i], "-c") == 0) {
      g.iterations.min_escape_iterations = ParseIntArg(argc, argv, i);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-w") == 0) {
      g.dimensions.w = ParseIntArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-h") == 0) {
      g.dimensions.h = ParseIntArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-g") == 0) {
      g.gamma_correction = ParseDoubleArg(argc, argv, i);
      i++;
      continue;
    }
    if (strcmp(argv[i], "-t") == 0) {
      g.seconds_to_run = ParseDoubleArg(argc, argv, i);
      i++;
      continue;
    }
    if (strcmp(argv[i], "--min-real") == 0) {
      g.dimensions.min_real = ParseDoubleArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    if (strcmp(argv[i], "--max-real") == 0) {
      g.dimensions.max_real = ParseDoubleArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    if (strcmp(argv[i], "--min-imag") == 0) {
      g.dimensions.min_imag = ParseDoubleArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    if (strcmp(argv[i], "--max-imag") == 0) {
      g.dimensions.max_imag = ParseDoubleArg(argc, argv, i);
      if (!RecomputePixelDeltas()) PrintUsage(argv[0]);
      i++;
      continue;
    }
    // Unrecognized argument, print the usage string.
    printf("Invalid argument: %s\n", argv[i]);
    PrintUsage(argv[0]);
  }
}

void SignalHandler(int signal_number) {
  g.quit_signal_received = 1;
  printf("Signal %d received, waiting for current pass to finish...\n",
    signal_number);
}

int main(int argc, char **argv) {
  memset(&g, 0, sizeof(g));
  g.output_image = DEFAULT_OUTPUT_NAME;
  g.iterations.max_escape_iterations = 100;
  g.iterations.min_escape_iterations = 20;
  g.iterations.samples_per_thread = SAMPLES_PER_THREAD;
  g.block_size = DEFAULT_BLOCK_SIZE;
  g.block_count = DEFAULT_BLOCK_COUNT;
  g.seconds_to_run = 10.0;
  g.gamma_correction = 1.0;
  SetDefaultCanvas();
  g.cuda_device = 0;
  ParseArguments(argc, argv);
  if (signal(SIGINT, SignalHandler) == SIG_ERR) {
    printf("Failed setting signal handler.\n");
    CleanupGlobals();
    return 1;
  }
  printf("Creating %dx%d image, %d samples per thread, %d max iterations.\n",
    g.dimensions.w, g.dimensions.h, g.iterations.samples_per_thread,
    g.iterations.max_escape_iterations);
  printf("Calculating image...\n");
  SetupCUDA();
  RenderImage();
  printf("Done! Saving image.\n");
  SaveImage();
  printf("Output image saved: %s\n", g.output_image);
  CleanupGlobals();
  return 0;
}
