#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
extern "C" {
#include <SDL2/SDL.h>
}

// The number of CUDA threads to use per block.
#define DEFAULT_BLOCK_SIZE (128)

// The number of iterations to record the paths of points that escape the set.
#define PATH_ITERATIONS (20000)

// This macro takes a hipError_t value and exits the program if it isn't equal
// to hipSuccess. (Calls the ErrorCheck function, defined later).
#define CheckCUDAError(val) (InternalCUDAErrorCheck((val), #val, __FILE__, __LINE__))

// Holds the boundaries and sizes of the fractal, in both pixels and numbers
typedef struct {
  // The width and height of the image in pixels.
  int w;
  int h;
  // The boundaries of the fractal.
  double min_real;
  double min_imag;
  double max_real;
  double max_imag;
  // The distance between pixels in the real and imaginary axes.
  double delta_real;
  double delta_imag;
} FractalDimensions;

// Tracks a single pair of points which escaped the mandelbrot set. These will
// be used as the start points of buddhabrot paths.
typedef struct {
  double real;
  double imag;
} EscapingPoint;

// Holds globals in a single namespace.
static struct {
  SDL_Window *window;
  SDL_Renderer *renderer;
  SDL_Texture *image;
  // The maximum number of iterations to run each point in the initial
  // mandelbrot calculation.
  int mandelbrot_iterations;
  // The number of iterations to track the paths of escaping points in the
  // buddhabrot.
  int buddhabrot_iterations;
  // The size and location of the fractal and output image.
  FractalDimensions dimensions;
  // Pointer to the device memory that will contain 0 if a point is in the set,
  // and 1 if it escapes the set.
  uint8_t *device_mandelbrot;
  // The host-side copy of the basic binary mandelbrot set.
  uint8_t *host_mandelbrot;
  // Lists of points which escape the mandelbrot set.
  EscapingPoint *host_escaping_points;
  EscapingPoint *device_escaping_points;
  // The number of points which escaped the mandelbrot set.
  int escaping_point_count;
  // The host and device buffers which contain the numbers of times an escaping
  // point's path crossed each point in the complex plane.
  uint32_t *device_buddhabrot;
  uint32_t *host_buddhabrot;
} g;

// If any globals have been initialized, this will free them. (Relies on
// globals being set to 0 at the start of the program)
static void CleanupGlobals(void) {
  if (g.renderer) SDL_DestroyRenderer(g.renderer);
  if (g.image) SDL_DestroyTexture(g.image);
  if (g.window) SDL_DestroyWindow(g.window);
  if (g.device_mandelbrot) hipFree(g.device_mandelbrot);
  if (g.host_mandelbrot) free(g.host_mandelbrot);
  if (g.host_escaping_points) free(g.host_escaping_points);
  if (g.device_escaping_points) hipFree(g.device_escaping_points);
  memset(&g, 0, sizeof(g));
}

// Returns the current time in seconds.
static double CurrentSeconds(void) {
  struct timespec ts;
  if (clock_gettime(CLOCK_REALTIME, &ts) != 0) {
    printf("Error getting time.\n");
    exit(1);
  }
  return ((double) ts.tv_sec) + (((double) ts.tv_nsec) / 1e9);
}

// Prints an error message and exits the program if the hipError_t value is
// not equal to hipSuccess. Generally, this will be called via the
// CheckCudaError macro.
static void InternalCUDAErrorCheck(hipError_t result, const char *fn,
    const char *file, int line) {
  if (result == hipSuccess) return;
  printf("CUDA error %d in %s, line %d (%s)\n", (int) result, file, line, fn);
  exit(1);
  CleanupGlobals();
}

// Sets up the SDL window and resources. Must be called after g.w and g.h have
// been set.
static void SetupSDL(void) {
  if (SDL_Init(SDL_INIT_EVERYTHING) < 0) {
    printf("SDL error %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.window = SDL_CreateWindow("Rendered image", SDL_WINDOWPOS_UNDEFINED,
    SDL_WINDOWPOS_UNDEFINED, g.dimensions.w, g.dimensions.h, SDL_WINDOW_SHOWN |
    SDL_WINDOW_RESIZABLE);
  if (!g.window) {
    printf("Error creating SDL window: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.renderer = SDL_CreateRenderer(g.window, -1, SDL_RENDERER_ACCELERATED);
  if (!g.renderer) {
    printf("Error creating SDL renderer: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  g.image = SDL_CreateTexture(g.renderer, SDL_PIXELFORMAT_RGBA8888,
    SDL_TEXTUREACCESS_STREAMING, g.dimensions.w, g.dimensions.h);
  if (!g.image) {
    printf("Failed creating SDL texture: %s\n", SDL_GetError());
    exit(1);
  }
}

// Allocates CUDA memory and calculates block/grid sizes. Must be called after
// g.w and g.h have been set.
static void SetupCUDA(void) {
  CheckCUDAError(hipFree(0));
  size_t buffer_size = g.dimensions.w * g.dimensions.h;
  CheckCUDAError(hipMalloc(&(g.device_mandelbrot), buffer_size));
  CheckCUDAError(hipMemset(g.device_mandelbrot, 0, buffer_size));
  g.host_mandelbrot = (uint8_t *) malloc(buffer_size);
  if (!g.host_mandelbrot) {
    printf("Failed allocating host mandelbrot buffer.\n");
    CleanupGlobals();
    exit(1);
  }
  memset(g.host_mandelbrot, 0, buffer_size);
  CheckCUDAError(hipMalloc(&(g.device_buddhabrot), buffer_size *
    sizeof(uint32_t)));
  CheckCUDAError(hipMemset(g.device_buddhabrot, 0, buffer_size *
    sizeof(uint32_t)));
  g.host_buddhabrot = (uint32_t *) malloc(buffer_size * sizeof(uint32_t));
  if (!g.host_buddhabrot) {
    printf("Failed allocating host buddhabrot buffer.\n");
    CleanupGlobals();
    exit(1);
  }
  memset(g.host_buddhabrot, 0, buffer_size * sizeof(uint32_t));
}

// A basic mandelbrot set calculator which sets each element in data to 1 if
// the point escapes within the given number of iterations.
__global__ void BasicMandelbrot(uint8_t *data, int iterations,
    FractalDimensions dimensions) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int row = index / dimensions.w;
  int col = index % dimensions.w;
  // This may cause some threads to diverge on the last block only
  if (row >= dimensions.h) return;
  double start_real = dimensions.min_real + dimensions.delta_real * col;
  double start_imag = dimensions.min_imag + dimensions.delta_imag * row;
  double current_real = start_real;
  double current_imag = start_imag;
  double magnitude_squared = (start_real * start_real) + (start_imag *
    start_imag);
  uint8_t escaped = 0;
  double tmp;
  int i;
  for (i = 0; i < iterations; i++) {
    if (magnitude_squared < 4) {
      tmp = (current_real * current_real) - (current_imag * current_imag) +
        start_real;
      current_imag = 2 * current_imag * current_real + start_imag;
      current_real = tmp;
      magnitude_squared = (current_real * current_real) + (current_imag *
        current_imag);
    } else {
      escaped = 1;
    }
  }
  data[row * dimensions.w + col] = escaped;
}

// After BasicMandelbrot has been completed, and host_mandelbrot has been
// filled in, this will allocate and populate both device_escaping_points and
// host_escaping_points.
static void GatherEscapingPoints(void) {
  int w = g.dimensions.w;
  int h = g.dimensions.h;
  int x, y;
  size_t points_size = 0;
  int points_added = 0;
  EscapingPoint *escaping_point = NULL;

  // First, get a count of the escaping points, so the correct amount of memory
  // can be allocated.
  int count = 0;
  for (y = 0; y < h; y++) {
    for (x = 0; x < w; x++) {
      if (g.host_mandelbrot[y * w + x]) count++;
    }
  }
  g.escaping_point_count = count;

  // Next, build the list of escaping points and copy it to GPU memory.
  points_size = count * sizeof(EscapingPoint);
  g.host_escaping_points = (EscapingPoint *) malloc(points_size);
  if (!g.host_escaping_points) {
    printf("Failed allocating space for escaping point list.\n");
    CleanupGlobals();
    exit(1);
  }
  CheckCUDAError(hipMalloc(&(g.device_escaping_points), points_size));
  for (y = 0; y < h; y++) {
    for (x = 0; x < w; x++) {
      if (!g.host_mandelbrot[y * w + x]) continue;
      escaping_point = g.host_escaping_points + points_added;
      escaping_point->real = ((double) x) * g.dimensions.delta_real +
        g.dimensions.min_real;
      escaping_point->imag = ((double) y) * g.dimensions.delta_imag +
        g.dimensions.min_imag;
      points_added++;
    }
  }
  CheckCUDAError(hipMemcpy(g.device_escaping_points, g.host_escaping_points,
    points_size, hipMemcpyHostToDevice));
}

// This kernel takes a list of points which escape the mandelbrot set, and, for
// each iteration of the point, increments its location in the data array.
__global__ void DrawBuddhabrot(EscapingPoint *points, int point_count,
    uint32_t *data, int iterations, FractalDimensions dimensions) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= point_count) return;
  int i;
  double start_real = points[index].real;
  double start_imag = points[index].imag;
  double current_real = start_real;
  double current_imag = start_imag;
  double tmp;
  int row, col;
  // This should only happen in the final block.
  if (index > point_count) return;
  for (i = 0; i < iterations; i++) {
    tmp = (current_real * current_real) - (current_imag * current_imag) +
      start_real;
    current_imag = 2 * current_real * current_imag + start_imag;
    current_real = tmp;
    row = (current_imag - dimensions.min_imag) / dimensions.delta_imag;
    col = (current_real - dimensions.min_real) / dimensions.delta_real;
    if ((row >= 0) && (row < dimensions.h) && (col >= 0) && (col <
      dimensions.w)) {
      data[row * dimensions.w + col]++;
    }
  }
}

// Renders the fractal image.
static void RenderImage(void) {
  int block_count;
  size_t data_size = g.dimensions.w * g.dimensions.h;
  double seconds;

  printf("Calculating initial mandelbrot set.\n");
  // First, draw the basic mandelbrot to get which points escape.
  block_count = (data_size / DEFAULT_BLOCK_SIZE) + 1;
  seconds = CurrentSeconds();
  BasicMandelbrot<<<block_count, DEFAULT_BLOCK_SIZE>>>(g.device_mandelbrot,
    g.mandelbrot_iterations, g.dimensions);
  CheckCUDAError(hipGetLastError());
  CheckCUDAError(hipMemcpy(g.host_mandelbrot, g.device_mandelbrot,
    data_size, hipMemcpyDeviceToHost));
  printf("Mandelbrot took %f seconds.\n", CurrentSeconds() - seconds);

  printf("Finding start points for buddhabrot.\n");
  GatherEscapingPoints();

  printf("Calculating buddhabrot.\n");
  block_count = (g.escaping_point_count / DEFAULT_BLOCK_SIZE) + 1;
  seconds = CurrentSeconds();
  DrawBuddhabrot<<<block_count, DEFAULT_BLOCK_SIZE>>>(g.device_escaping_points,
    g.escaping_point_count, g.device_buddhabrot, g.buddhabrot_iterations,
    g.dimensions);
  CheckCUDAError(hipGetLastError());
  CheckCUDAError(hipMemcpy(g.host_buddhabrot, g.device_buddhabrot,
    data_size * sizeof(uint32_t), hipMemcpyDeviceToHost));
  printf("Buddhabrot took %f seconds.\n", CurrentSeconds() - seconds);
}

static double GetColorScale(void) {
  // TODO: Figure out a color scaling system that works:
  //  - Highest value should still be 255.
  //  - Lowest value should be some pale gray
  //  - Should have a lograthmic curve that amplifies low values a lot
  return 16;
}

static uint8_t Clamp(double v) {
  if (v <= 0) return 0;
  if (v >= 255) return 255;
  return (uint8_t) v;
}

// Copies data from the host-side data buffer to the texture drawn to the SDL
// window.
static void UpdateDisplayedImage(void) {
  int x, y;
  uint8_t *image_pixels;
  int image_pitch;
  int to_skip_per_row;
  uint8_t color_value;
  double color_scale = GetColorScale();
  uint32_t *host_data = g.host_buddhabrot;
  if (SDL_LockTexture(g.image, NULL, (void **) (&image_pixels), &image_pitch)
    < 0) {
    printf("Error locking SDL texture: %s\n", SDL_GetError());
    CleanupGlobals();
    exit(1);
  }
  // Abide by the image pitch, and skip unaffected bytes in each row.
  // (image_pitch should usually be equal to g.w * 4 anyway).
  to_skip_per_row = image_pitch - (g.dimensions.w * 4);
  for (y = 0; y < g.dimensions.h; y++) {
    for (x = 0; x < g.dimensions.w; x++) {
      color_value = Clamp(color_scale * (*host_data));
      // The byte order is ABGR
      image_pixels[0] = 0xff;
      image_pixels[1] = color_value;
      image_pixels[2] = color_value;
      image_pixels[3] = color_value;
      image_pixels += 4;
      host_data++;
    }
    image_pixels += to_skip_per_row;
  }
  SDL_UnlockTexture(g.image);
}

// Runs the main loop to display the SDL window. This will return when SDL
// detects an exit event.
static void SDLWindowLoop(void) {
  SDL_Event event;
  int quit = 0;
  // Update the display once every 30 ms (not really necessary for now, while
  // it doesn't change...
  while (!quit) {
    while (SDL_PollEvent(&event)) {
      if (event.type == SDL_QUIT) {
        quit = 1;
        break;
      }
    }
    UpdateDisplayedImage();
    if (SDL_RenderCopy(g.renderer, g.image, NULL, NULL) < 0) {
      printf("Error rendering image: %s\n", SDL_GetError());
      CleanupGlobals();
      exit(1);
    }
    SDL_RenderPresent(g.renderer);
    usleep(20000);
  }
}

// Sets the resolution, scaling the complex boundaries to maintain an aspect
// ratio.
static void SetResolution(int width, int height) {
  FractalDimensions *dims = &(g.dimensions);
  double ratio = ((double) height) / ((double) width);
  // The horizontal width for which the complex plane is shown.
  double real_width = 4.0;
  double imag_width = real_width * ratio;
  dims->w = width;
  dims->h = height;
  dims->min_real = -(real_width / 2.0);
  dims->max_real = dims->min_real + real_width;
  dims->min_imag = -(imag_width / 2.0);
  dims->max_imag = dims->min_imag + imag_width;
  dims->delta_imag = imag_width / ((double) height);
  dims->delta_real = real_width / ((double) width);
}

int main(int argc, char **argv) {
  memset(&g, 0, sizeof(g));
  SetResolution(3840, 2400);
  g.mandelbrot_iterations = 100;
  g.buddhabrot_iterations = 20000;
  printf("Calculating image...\n");
  SetupCUDA();
  RenderImage();
  printf("Done!\n");
  SetupSDL();
  SDLWindowLoop();
  CleanupGlobals();
  return 0;
}
